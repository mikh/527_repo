
#include <hip/hip_runtime.h>
#include <math.h>
#include <cstdio>
#include <cstdlib>
#include <time.h>

#define GIG 1000000000
#define NANO_TO_MILLI 1000000
#define CPG 3.6         // Cycles per GHz -- Adjust to your computer

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

const int MATRIX_SIZE = 2000;
const int NUM_BLOCKS = 1;
const int THREADS_PER_BLOCK_X = 16;
const int THREADS_PER_BLOCK_Y = 16;
const int SOR_ITERATIONS = 2000;
const int OMEGA = 1;

#define ALLOCATE_AND_INIT
#define TRANSFER_TO_GPU
#define LAUNCH_KERNEL
#define TRANSFER_RESULTS
#define COMPUTE_CPU_RESULTS
#define COMPARE_RESULTS
#define FREE_MEMORY
#define GPU_TIMING
#define CPU_TIMING
#define DEBUG_PRINT
#define WRITE_2D_ARRAYS

void initialize_array_2D(float **A, int len, int seed);

__global__ void kernel_SOR_internal(float *A, int omega, int N_x, int N_y){
	int i, j = 0;
	//different divisions needed for group of threads etc.
	int xx = blockIdx.x * blockDim.x + threadIdx.x;
	int yy = blockIdx.x * blockDim.y + threadIdx.y;
	float phi;
	for(i = MATRIX_SIZE/THREADS_PER_BLOCK_X*xx; i < MATRIX_SIZE/THREADS_PER_BLOCK_X*(xx+1); i++){
		for(j = MATRIX_SIZE/THREADS_PER_BLOCK_Y*yy; j < MATRIX_SIZE/THREADS_PER_BLOCK_Y*(yy+1); j++){
			if(i > 0 && i < (N_x-1) && j > 0 && j < (N_y-1)){
				phi = A[i*MATRIX_SIZE + j] - .25*((A[(i-1)*MATRIX_SIZE + j] + A[(i+1)*MATRIX_SIZE+j]) + (A[i*MATRIX_SIZE + (j-1)] + A[i*MATRIX_SIZE+(j+1)]));
				A[i*MATRIX_SIZE+j] = abs(A[i*MATRIX_SIZE+j] - (phi*omega));
			}
		}
	}
}

void SOR_internal_sequential(float **A, int omega, int xx, int yy, int N_x, int N_y){
	float phi;
	if(xx > 0 && xx < N_x-1 && yy > 0 && yy < N_y-1){
		phi = A[xx][yy] - .25*((A[xx-1][yy] + A[xx+1][yy]) + (A[xx][yy-1] + A[xx][yy+1]));
		A[xx][yy] = abs(A[xx][yy] - (phi*omega));
	}
}

void write_2d_array_to_file(float **A, int N_x, int N_y, char *filename){
	int i, j;
	FILE *f = fopen(filename, "w");
	for(i = 0; i < N_x; i++){
		for(j = 0; j < N_y-1; j++){
			fprintf(f,"%f, ",A[i][j]);
		}
		fprintf(f, "%f\n", A[i][N_y-1]);
	}
	fclose(f);
}

int main(int argc, char **argv){

	//loop variables
	int i, j, k, errors = 0;

	//timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	struct timespec diff(struct timespec start, struct timespec end);
	struct timespec time1, time2, elapsed_cpu;

	//array dimensions
	dim3 dimGrid(NUM_BLOCKS,1,1);
	dim3 dimBlock(THREADS_PER_BLOCK_X,THREADS_PER_BLOCK_Y,1);
	
	//Arrays on GPU global memory
	float *g_A;

	//Arrays on host memory	
	float **h_A;
	float **h_A_test;

	//error file init
	FILE *f = fopen("mismatches.txt", "w");
	if(f == NULL){
		printf("Error opening file!\n");
		exit(1);
	}

#ifdef DEBUG_PRINT
	printf("init done\n");
#endif

	//Allocate arrays on GPU memory
#ifdef ALLOCATE_AND_INIT
	CUDA_SAFE_CALL(hipMalloc((void **) &g_A, MATRIX_SIZE*MATRIX_SIZE*sizeof(float)));

	//Allocate arrays on host memory
	h_A = (float**) malloc(MATRIX_SIZE * sizeof(float*));
	h_A_test = (float**) malloc(MATRIX_SIZE * sizeof(float*));
	for(i = 0; i < MATRIX_SIZE; i++){
		h_A[i] = (float*) malloc(MATRIX_SIZE * sizeof(float));
		h_A_test[i] = (float*) malloc(MATRIX_SIZE * sizeof(float));
	}

	//initialize host arrays
	initialize_array_2D(h_A, MATRIX_SIZE, 2453);
	initialize_array_2D(h_A_test, MATRIX_SIZE, 2453);
#endif 

#ifdef GPU_TIMING
	//create cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//record event on default stream
	hipEventRecord(start, 0);
#endif

#ifdef DEBUG_PRINT
	printf("all init done\n");
#endif
	//transfer array to GPU memory
#ifdef TRANSFER_TO_GPU
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipMemcpy(&g_A[i*MATRIX_SIZE], h_A[i], MATRIX_SIZE, hipMemcpyHostToDevice));
	}
#endif

	//launch the kernel
#ifdef LAUNCH_KERNEL
	for(i = 0; i < SOR_ITERATIONS; i++){
		kernel_SOR_internal<<<dimGrid, dimBlock>>>(g_A, OMEGA, MATRIX_SIZE, MATRIX_SIZE);
	}
#endif



	//check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());

#ifdef DEBUG_PRINT
	printf("kernel run\n");
#endif

	//transfer results back to host
#ifdef TRANSFER_RESULTS
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipMemcpy(h_A[i], &g_A[i*MATRIX_SIZE], MATRIX_SIZE, hipMemcpyDeviceToHost));
	}
#endif
	//stop and destroy the timer
#ifdef GPU_TIMING
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif

#ifdef DEBUG_PRINT
	printf("results transfered\n");
#endif
	//compute results on host

#ifdef CPU_TIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
#endif

#ifdef COMPUTE_CPU_RESULTS
	for(i = 0; i < SOR_ITERATIONS; i++){
		for(j = 0; j < MATRIX_SIZE; j++){
			for(k = 0; k < MATRIX_SIZE; k++){
				SOR_internal_sequential(h_A_test, OMEGA, j, k, MATRIX_SIZE, MATRIX_SIZE);
			}
		}
	}
#endif

#ifdef CPU_TIMING
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed_cpu = diff(time1,time2);
	printf("\nCPU time: %f(msec)\n", (float)(((double)GIG*elapsed_cpu.tv_sec + elapsed_cpu.tv_nsec)/(double)NANO_TO_MILLI));
#endif


#ifdef DEBUG_PRINT
	printf("results computed on CPU\n");
#endif
	//compare results

#ifdef COMPARE_RESULTS	
	for(i = 0; i < MATRIX_SIZE; i++){
		for(j = 0; j < MATRIX_SIZE; j++){
			if(h_A[i][j] != h_A_test[i][j]){
				errors++;
				fprintf(f, "Mismatch at [%d,%d] GPU = %f CPU = %f\n", i, j, h_A[i][j], h_A_test[i][j]);
			}
		}
	}
#endif

#ifdef DEBUG_PRINT
	printf("results checked\n");
#endif

#ifdef WRITE_2D_ARRAYS
	write_2d_array_to_file(h_A, MATRIX_SIZE, MATRIX_SIZE, "GPU_output.txt");
	write_2d_array_to_file(h_A_test, MATRIX_SIZE, MATRIX_SIZE, "CPU_output.txt");
#endif

	//errors
	printf("Found %d errors\n", errors);
	fclose(f);

	//free up memory
#ifdef FREE_MEMORY
	CUDA_SAFE_CALL(hipFree(g_A));
	for(i = 0; i < MATRIX_SIZE; i++){
		free(h_A[i]);
		free(h_A_test[i]);
	}
	free(h_A);
	free(h_A_test);
#ifdef DEBUG_PRINT
	printf("arrays freed\n");
#endif
#endif

	return (float)0;
}

void initialize_array_2D(float **A, int len, int seed){
	int i, j;
	float randNum;
	srand(seed);

	for(i = 0; i < len; i++){
		for(j = 0; j < len; j++){
			randNum = (float) rand();
			A[i][j] = randNum;
		}
	}
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
