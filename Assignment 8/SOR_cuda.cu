#include "hip/hip_runtime.h"
#include <math.h>

const int MATRIX_SIZE = 2000;

void initialize_array_2D(float **A, int len, int seed);

__global__ void kernel_SOR_internal(float **A, int omega, int N_x, int N_y){
	int xx = blockIdx.x * blockDim.x + threadIdx.x;
	int yy = blockIdy.y * blockDim.y + threadIny.y;
	float phi;

	if(xx > 0 && xx < (N_x-1) && yy > 0 && yy < (N_y-1)){
		phi = A[xx][yy] - .25((A[xx-1][yy] + A[xx+1][yy]) + (A[xx][yy-1] + A[xx][yy+1]))
		A[xx][yy] = abs(A[xx][yy] - (phi*omega));
	}
}

int main(int argc, char **argv){

	int i;
	
	//Arrays on GPU global memory
	float **g_A;

	//Arrays on host memory	
	float **h_A;


	//Allocate arrays on GPU memory
	CUDA_SAFE_CALL(hipMalloc((void **)&g_A, MATRIX_SIZE * sizeof(float*)));
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipMalloc((void**)&(g_A[i]), MATRIX_SIZE * sizeof(float)));
	}

	//Allocate arrays on host memory
	h_A = (float**) malloc(MATRIX_SIZE * sizeof(float*));
	for(i = 0; i < MATRIX_SIZE; i++){
		h_A[i] = (float*) malloc(MATRIX_SIZE * sizeof(float));
	}

	//initialize host arrays
	initializeArray1D(h_A, arrLen, 2453);
}

void initialize_array_2D(float **A, int len, int seed){
	int i, j;
	float randNum;
	srand(seed);

	for(i = 0; i < len; i++){
		for(j = 0; j < len; j++){
			randNum = (float) rand();
			A[i][j] = randNum;
		}
	}
}

