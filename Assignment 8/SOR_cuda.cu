#include "hip/hip_runtime.h"
#include <math.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

const int MATRIX_SIZE = 2000;
const int NUM_BLOCKS = 1;
const int THREADS_PER_BLOCK_X = 16;
const int THREADS_PER_BLOCK_Y = 16;
const int SOR_ITERATIONS = 2000;
const int OMEGA = 1;

void initialize_array_2D(float **A, int len, int seed);

__global__ void kernel_SOR_internal(float **A, int omega, int N_x, int N_y){
	int i, j;
	//different divisions needed for group of threads etc.
	int xx = blockIdx.x * blockDim.x + threadIdx.x;
	int yy = blockIdy.y * blockDim.y + threadIny.y;
	float phi;
	for(i = MATRIX_SIZE/THREADS_PER_BLOCK_X*xx; i < MATRIX_SIZE/THREADS_PER_BLOCK_X*(xx+1); i++){
		for(j = MATRIX_SIZE/THREADS_PER_BLOCK_Y*yy; j < MATRIX_SIZE/THREADS_PER_BLOCK_Y*(yy+1); j++){
			if(i > 0 && i < (N_x-1) && j > 0 && j < (N_y-1)){
				phi = A[i][j] - .25((A[i-1][j] + A[i+1][j]) + (A[i][j-1] + A[i][j+1]))
				A[i][j] = abs(A[i][j] - (phi*omega));
			}
		}
	}
}

int main(int argc, char **argv){

	int i;
	dim3 dimGrid(NUM_BLOCKS,1,1);
	dim3 dimBlock(THREADS_PER_BLOCK_X,THREADS_PER_BLOCK_Y,1);
	//Arrays on GPU global memory
	float **g_A;

	//Arrays on host memory	
	float **h_A;


	//Allocate arrays on GPU memory
	CUDA_SAFE_CALL(hipMalloc((void **)&g_A, MATRIX_SIZE * sizeof(float*)));
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipMalloc((void**)&(g_A[i]), MATRIX_SIZE * sizeof(float)));
	}

	//Allocate arrays on host memory
	h_A = (float**) malloc(MATRIX_SIZE * sizeof(float*));
	for(i = 0; i < MATRIX_SIZE; i++){
		h_A[i] = (float*) malloc(MATRIX_SIZE * sizeof(float));
	}

	//initialize host arrays
	initialize_array_2D(h_A, MATRIX_SIZE, 2453);



	//create cuda events

	//record event on default stream


	//transfer array to GPU memory
	CUDA_SAFE_CALL(hipMemcpy(g_A, h_A, MATRIX_SIZE, hipMemcpyHostToDevice));
	for(i = 0; i < MATRIX_SIZE; i++){	//NOTE: might have to use pointer stuff here
		CUDA_SAFE_CALL(hipMemcpy(g_A[i], h_A[i], MATRIX_SIZE, hipMemcpyHostToDevice));
	}

	//launch the kernel
	for(i = 0; i < SOR_ITERATIONS; i++){
		kernel_SOR_internal<<<dimGrid, dimBlock>>>(g_A, OMEGA, MATRIX_SIZE, MATRIX_SIZE);
	}

	//check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());

	//transfer results back to host
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, allocSize, hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipMemcpy(h_A, g_A, MATRIX_SIZE, hipMemcpyDeviceToHost));
	for(i = 0; i < MATRIX_SIZE; i++){	//NOTE: might have to use pointer stuff here
		CUDA_SAFE_CALL(hipMemcpy(h_A[i], g_A[i], MATRIX_SIZE, hipMemcpyDeviceToHost));
	}

	//stop and destroy the timer

	//compute results on host

	//compare results

	//errors

	//free up memory
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipFree(g_A[i]));
	}
	CUDA_SAFE_CALL(hipFree(g_A));

	for(i = 0; i < MATRIX_SIZE; i++){
		free(h_A[i]);
	}
	free(h_A);

	return 0;
}

void initialize_array_2D(float **A, int len, int seed){
	int i, j;
	float randNum;
	srand(seed);

	for(i = 0; i < len; i++){
		for(j = 0; j < len; j++){
			randNum = (float) rand();
			A[i][j] = randNum;
		}
	}
}

