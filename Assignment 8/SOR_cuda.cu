
#include <hip/hip_runtime.h>
#include <math.h>
#include <cstdio>
#include <cstdlib>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

const int MATRIX_SIZE = 100;
const int NUM_BLOCKS = 1;
const int THREADS_PER_BLOCK_X = 16;
const int THREADS_PER_BLOCK_Y = 16;
const int SOR_ITERATIONS = 3;
const int OMEGA = 1;

#define ALLOCATE_AND_INIT
#define TRANSFER_TO_GPU
#define LAUNCH_KERNEL
#define TRANSFER_RESULTS
#define COMPUTE_CPU_RESULTS
#define COMPARE_RESULTS
#define FREE_MEMORY

void initialize_array_2D(float **A, int len, int seed);

__global__ void kernel_SOR_internal(float *A, int omega, int N_x, int N_y){
	int i, j = 0;
	//different divisions needed for group of threads etc.
	int xx = blockIdx.x * blockDim.x + threadIdx.x;
	int yy = blockIdx.x * blockDim.y + threadIdx.y;
	float phi;
	for(i = MATRIX_SIZE/THREADS_PER_BLOCK_X*xx; i < MATRIX_SIZE/THREADS_PER_BLOCK_X*(xx+1); i++){
		for(j = MATRIX_SIZE/THREADS_PER_BLOCK_Y*yy; j < MATRIX_SIZE/THREADS_PER_BLOCK_Y*(yy+1); j++){
			if(i > 0 && i < (N_x-1) && j > 0 && j < (N_y-1)){
				phi = A[i*MATRIX_SIZE + j] - .25*((A[(i-1)*MATRIX_SIZE + j] + A[(i+1)*MATRIX_SIZE+j]) + (A[i*MATRIX_SIZE + (j-1)] + A[i*MATRIX_SIZE+(j+1)]));
				A[i*MATRIX_SIZE+j] = abs(A[i*MATRIX_SIZE+j] - (phi*omega));
			}
		}
	}
}

void SOR_internal_sequential(float **A, int omega, int xx, int yy, int N_x, int N_y){
	float phi;
	if(xx > 0 && xx < N_x-1 && yy > 0 && yy < N_y-1){
		phi = A[xx][yy] - .25*((A[xx-1][yy] + A[xx+1][yy]) + (A[xx][yy-1] + A[xx][yy+1]));
		A[xx][yy] = abs(A[xx][yy] - (phi*omega));
	}
}

void write_2d_array_to_file(float **A, int N_x, int N_y, char *filename){
	int i, j;
	FILE *f = fopen(filename, "w");
	for(i = 0; i < N_x; i++){
		for(j = 0; j < N_y-1; j++){
			fprintf(f,"%f, ",A[i][j]);
		}
		fprintf(f, "%f\n", A[i][N_y-1]);
	}
	fclose(f);
}

int main(int argc, char **argv){

	int i, j, k, errors = 0;
	dim3 dimGrid(NUM_BLOCKS,1,1);
	dim3 dimBlock(THREADS_PER_BLOCK_X,THREADS_PER_BLOCK_Y,1);
	//Arrays on GPU global memory
	float *g_A;

	//Arrays on host memory	
	float **h_A;
	float **h_A_test;


	FILE *f = fopen("mismatches.txt", "w");
	if(f == NULL){
		printf("Error opening file!\n");
		exit(1);
	}

	printf("init done\n");
	//Allocate arrays on GPU memory
#ifdef ALLOCATE_AND_INIT
	CUDA_SAFE_CALL(hipMalloc((void **) &g_A, MATRIX_SIZE*MATRIX_SIZE*sizeof(float)));

	//Allocate arrays on host memory
	h_A = (float**) malloc(MATRIX_SIZE * sizeof(float*));
	h_A_test = (float**) malloc(MATRIX_SIZE * sizeof(float*));
	for(i = 0; i < MATRIX_SIZE; i++){
		h_A[i] = (float*) malloc(MATRIX_SIZE * sizeof(float));
		h_A_test[i] = (float*) malloc(MATRIX_SIZE * sizeof(float));
	}

	//initialize host arrays
	initialize_array_2D(h_A, MATRIX_SIZE, 2453);
	initialize_array_2D(h_A_test, MATRIX_SIZE, 2453);
#endif 

	//create cuda events

	//record event on default stream

	printf("all init done\n");
	//transfer array to GPU memory
#ifdef TRANSFER_TO_GPU
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipMemcpy(&g_A[i*MATRIX_SIZE], h_A[i], MATRIX_SIZE, hipMemcpyHostToDevice));
	}
#endif

	//launch the kernel
#ifdef LAUNCH_KERNEL
	for(i = 0; i < SOR_ITERATIONS; i++){
		kernel_SOR_internal<<<dimGrid, dimBlock>>>(g_A, OMEGA, MATRIX_SIZE, MATRIX_SIZE);
	}
#endif



	//check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());

	printf("kernel run\n");

	//transfer results back to host
#ifdef TRANSFER_RESULTS
	for(i = 0; i < MATRIX_SIZE; i++){
		CUDA_SAFE_CALL(hipMemcpy(h_A[i], &g_A[i*MATRIX_SIZE], MATRIX_SIZE, hipMemcpyDeviceToHost));
	}
#endif
	//stop and destroy the timer

	printf("results transfered\n");
	//compute results on host
#ifdef COMPUTE_CPU_RESULTS
	for(i = 0; i < SOR_ITERATIONS; i++){
		for(j = 0; j < MATRIX_SIZE; j++){
			for(k = 0; k < MATRIX_SIZE; k++){
				SOR_internal_sequential(h_A_test, OMEGA, j, k, MATRIX_SIZE, MATRIX_SIZE);
			}
		}
	}
#endif

	printf("results computed on CPU\n");
	//compare results

#ifdef COMPARE_RESULTS	
	for(i = 0; i < MATRIX_SIZE; i++){
		for(j = 0; j < MATRIX_SIZE; j++){
			if(h_A[i][j] != h_A_test[i][j]){
				errors++;
				fprintf(f, "Mismatch at [%d,%d] GPU = %f CPU = %f\n", i, j, h_A[i][j], h_A_test[i][j]);
			}
		}
	}
#endif

	printf("results checked\n");
	//write_2d_array_to_file(h_A, MATRIX_SIZE, MATRIX_SIZE, "GPU_output.txt");
	//write_2d_array_to_file(h_A_test, MATRIX_SIZE, MATRIX_SIZE, "CPU_output.txt");

	//errors
	printf("Found %d errors\n", errors);
	fclose(f);

	//free up memory
#ifdef FREE_MEMORY
	CUDA_SAFE_CALL(hipFree(g_A));
	for(i = 0; i < MATRIX_SIZE; i++){
		free(h_A[i]);
		free(h_A_test[i]);
	}
	free(h_A);
	free(h_A_test);
	printf("arrays freed\n");
#endif

	return (float)0;
}

void initialize_array_2D(float **A, int len, int seed){
	int i, j;
	float randNum;
	srand(seed);

	for(i = 0; i < len; i++){
		for(j = 0; j < len; j++){
			randNum = (float) rand();
			A[i][j] = randNum;
		}
	}
}

