
#include <hip/hip_runtime.h>
#include <math.h>
#include <cstdio>
#include <cstdlib>
#include <time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define GIG 1000000000
#define NANO_TO_MILLI 1000000
#define CPG 3.6         // Cycles per GHz -- Adjust to your computer

#define NUM_THREADS_PER_BLOCK 	16
#define NUM_BLOCKS 				16
#define PRINT_TIME 				1
#define SM_ARR_LEN				50000
#define TOL						1e-6

#define NN 100		//matrix size

#define THREADS_PER_BLOCK 16

#define THREADS_PER_BLOCK_Y 16
#define NUM_BLOCKS_X 16
#define NUM_BLOCKS_Y 16


#define IMUL(a, b) __mul24(a, b)



void initialize_array(float *A, int len, int seed);
void write_2d_array_to_file(float *A, char *filename);
struct timespec diff(struct timespec start, struct timespec end);

__global__ void kernel_MMM(float *A, float *B, float *C, int N){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int k;
	float sum = 0;

	if(i >= 0 && i < N && j >= 0 && j < N){
		for(k = 0; k < N; k++){
			sum += A[i*N+k] * B[k*N+j];
		}
		C[i*N+j] = sum;
	}
}

int main(int argc, char **argv){
	// GPU Timing variables
	int i, j, k;
	hipEvent_t start_i, start_o, stop_i, stop_o;
	float elapsed_gpu_internal, elapsed_gpu_with_copy;
	float max_difference, min_difference, average_difference, difference;
	struct timespec diff(struct timespec start, struct timespec end);
	struct timespec time1, time2, elapsed_cpu;


	//Arrays on GPU global memory
	float *g_A, *g_B, *g_C;

	//arrays on host memory
	float *h_A, *h_B, *h_C, *h_C_control;

	//Allocate arrays on GPU memory
	printf("Allocating arrays\n");
	CUDA_SAFE_CALL(hipMalloc((void **) &g_A, NN*NN*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void **) &g_B, NN*NN*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void **) &g_C, NN*NN*sizeof(float)));

	//Allocate arrays on host memory
	h_A = (float*) malloc(NN*NN*sizeof(float));
	h_B = (float*) malloc(NN*NN*sizeof(float));
	h_C = (float*) malloc(NN*NN*sizeof(float));
	h_C_control = (float*) malloc(NN*NN*sizeof(float));

	//initialize host arrays
	printf("Initializing host arrays\n");
	initialize_array(h_A, NN*NN, 1000);
	initialize_array(h_B, NN*NN, 2000);

	//create cuda events
	printf("Starting outer cuda timing\n");
	hipEventCreate(&start_o);
	hipEventCreate(&stop_o);
	//record event on default stream
	hipEventRecord(start_o, 0);

	//transfer array to GPU memory
	printf("Transfering arrays to GPU\n");
	CUDA_SAFE_CALL(hipMemcpy(g_A, h_A, NN*NN, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(g_B, h_B, NN*NN, hipMemcpyHostToDevice));

	//Launch the kernel
	//NN*NN/256 = # of blocks

	dim3 dimGrid(NN/THREADS_PER_BLOCK,NN/THREADS_PER_BLOCK);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK);

	//launch the kernel
	printf("Starting outer cuda timing\n");
	hipEventCreate(&start_i);
	hipEventCreate(&stop_i);
	//record event on default stream
	hipEventRecord(start_i, 0);

	printf("Running kernel\n");
	kernel_MMM<<<dimGrid, dimBlock>>>(g_A, g_B, g_C, NN);
	hipEventRecord(stop_i,0);
	hipEventSynchronize(stop_i);
	hipEventElapsedTime(&elapsed_gpu_internal, start_i, stop_i);
	hipEventDestroy(start_i);
	hipEventDestroy(stop_i);

	//check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());

	//transfer results back to host
	printf("Copying cuda results to host\n");
	CUDA_SAFE_CALL(hipMemcpy(h_C, g_C, NN*NN, hipMemcpyDeviceToHost));

	//stop and destroy the timer
	hipEventRecord(stop_o,0);
	hipEventSynchronize(stop_o);
	hipEventElapsedTime(&elapsed_gpu_with_copy, start_o, stop_o);
	hipEventDestroy(start_o);
	hipEventDestroy(stop_o);


	//compute result on host
	printf("Computing MMM on host\n");
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

	for(i = 0; i < NN; i++){
		for(j = 0; j < NN; j++){
			h_C_control[i*NN+j] = 0;
			for(k = 0; k < NN; k++){
				h_C_control[i*NN+j] += (h_A[i*NN+k] * h_B[k*NN + j]);
			}
		}
	}

	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	elapsed_cpu = diff(time1, time2);

	//write arrays to file
	printf("Writing arrays to file\n");
	write_2d_array_to_file(h_C, "gpu_MMM.txt");
	write_2d_array_to_file(h_C_control, "cpu_MMM.txt");

	//compare the results
	printf("Comparing results\n");
	max_difference = 0;
	min_difference = NN*NN;
	average_difference = 0;

	for(i = 0; i < NN; i++){
		for(j = 0; j < NN; j++){
			difference = abs(h_C[i*NN+j] - h_C_control[i*NN+j]);
			if(difference > max_difference)
				max_difference = difference;
			if(difference < min_difference)
				min_difference = difference;
			average_difference += difference;
		}
	}
	average_difference /= (float)(NN*NN);

	//free memory
	printf("Freeing memory\n");
	CUDA_SAFE_CALL(hipFree(g_A));
	CUDA_SAFE_CALL(hipFree(g_B));
	CUDA_SAFE_CALL(hipFree(g_C));
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_control);

	//print results
	printf("Printing results\n");
	printf("\nGPU outer loop time: %f (msec)\n", elapsed_gpu_with_copy);
	printf("\nGPU inner loop time: %f (msec)\n", elapsed_gpu_internal);
	printf("\nCPU time: %f(msec)\n", (float)(((double)GIG*elapsed_cpu.tv_sec + elapsed_cpu.tv_nsec)/(double)NANO_TO_MILLI));
	printf("Max difference = %f, Min difference = %f, Average difference = %f\n", max_difference, min_difference, average_difference);

	return 0;
}

void initialize_array(float *A, int len, int seed){
	int i;
	float randNum;

	srand(seed);

	for(i = 0; i < len; i++){
		randNum = (float) rand();
		A[i] = randNum;
	}
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}

void write_2d_array_to_file(float *A, char *filename){
	int i;
	FILE *f = fopen(filename, "w");
	for(i = 0; i < NN*NN; i++){
		fprintf(f, "%.0f, ", A[i]);
		if(i%NN == 0)
			fprintf(f, "\n");
	}
	fclose(f);
}